
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hiprand/hiprand.h>
#include <hipblas.h>

// Define some error checking macros.
#define cudaErrCheck(stat) { cudaErrCheck_((stat), __FILE__, __LINE__); }
void cudaErrCheck_(hipError_t stat, const char *file, int line) {
	if (stat != hipSuccess) {
		fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(stat), file, line);
	}
}

#define cublasErrCheck(stat) { cublasErrCheck_((stat), __FILE__, __LINE__); }
void cublasErrCheck_(hipblasStatus_t stat, const char *file, int line) {
	if (stat != HIPBLAS_STATUS_SUCCESS) {
		fprintf(stderr, "cuBLAS Error: %d %s %d\n", stat, file, line);
	}
}

#define curandErrCheck(stat) { curandErrCheck_((stat), __FILE__, __LINE__); }
void curandErrCheck_(hiprandStatus_t stat, const char *file, int line) {
	if (stat != HIPRAND_STATUS_SUCCESS) {
		fprintf(stderr, "cuRand Error: %d %s %d\n", stat, file, line);
	}
}


#include <mma.h>
using namespace nvcuda;

// Must be multiples of 16 for wmma code to work
#define MATRIX_M 16 // 12 x 4
#define MATRIX_N 16*12*12*12 // 16x12x12x12
//#define MATRIX_N 16 // 16x12x12x12
#define MATRIX_K 16 // MATRIX_M



// The only dimensions currently supported by WMMA
const int WMMA_M = 16;
const int WMMA_N = 16;
const int WMMA_K = 16;


// Performs an MxNxK GEMM (C=alpha*A*B + beta*C) assuming:
//  1) Matrices are packed in memory.
//  2) M, N and K are multiples of 16. 
//  3) Neither A nor B are transposed.
// Note: This is NOT a high performance example but is for demonstration purposes only
//       For a high performance code please use the GEMM provided in cuBLAS.
__global__ void wmma_example(half *a, half *b, float *c, int M, int N, int K, float alpha, float beta) {

	extern __shared__ float4 sm[];
	
	int global_n = blockIdx.x*blockDim.x+threadIdx.x;

	half* sm_a = ((half*)sm)+16*16*4;
	half* sm_b = sm_a + M*K;
	half* sm_c = sm_b + M*K;

	wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::col_major> a_frag;
  wmma::fragment<wmma::matrix_b, 16, 16, 16, half, wmma::col_major> b_frag;
  wmma::fragment<wmma::accumulator, 16, 16, 16, half> c_frag;

  // Initialize the output to zero
  wmma::fill_fragment(c_frag, (half)0.0f);

	if(threadIdx.x == 0){
		for(int k = 0; k < blockDim.y; k++){
			sm_a[threadIdx.y*blockDim.y+k] = a[threadIdx.y*blockDim.y+k];
		}
	}

	__syncthreads();

	sm_b[threadIdx.y*blockDim.x+threadIdx.x] = b[threadIdx.y*blockDim.x*gridDim.x+global_n];
	
	__syncthreads();

  // Load the inputs
  wmma::load_matrix_sync(a_frag, sm_a, 16);
  wmma::load_matrix_sync(b_frag, sm_b, 16);

  // Perform the matrix multiplication
  wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);

  // Store the output
  wmma::store_matrix_sync(sm_c, c_frag, 16, wmma::mem_col_major);

	__syncthreads();
	
	c[threadIdx.y*blockDim.x*gridDim.x+global_n] = sm_c[threadIdx.y*blockDim.x+threadIdx.x];

}

__global__ void convertFp32ToFp16 (half *out, float *in, int n) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx < n) {
		out[idx] = in[idx];
	}
}

int main(int argc, char* argv[]) {
	float *a_fp32;
	float *b_fp32;
	half *a_fp16;
	half *b_fp16;

	float *c;
	float *c_cublas;
	float *c_wmma;

	float *c_host_cublas;
	float *c_host_wmma;

	hiprandGenerator_t gen;
	hipblasHandle_t cublasHandle;

	hipEvent_t startWMMA;
	hipEvent_t stopWMMA;

	hipEvent_t startcublas;
	hipEvent_t stopcublas;

	cudaErrCheck(hipEventCreate(&startWMMA));
	cudaErrCheck(hipEventCreate(&stopWMMA));

	cudaErrCheck(hipEventCreate(&startcublas));
	cudaErrCheck(hipEventCreate(&stopcublas));


	cublasErrCheck(hipblasCreate(&cublasHandle));

	// Use tensor cores
	cublasErrCheck(hipblasSetMathMode(cublasHandle, HIPBLAS_TENSOR_OP_MATH));

	cudaErrCheck(hipMalloc((void**)&a_fp32, MATRIX_M * MATRIX_K * sizeof(float)));
	cudaErrCheck(hipMalloc((void**)&b_fp32, MATRIX_K * MATRIX_N * sizeof(float)));
	cudaErrCheck(hipMalloc((void**)&a_fp16, MATRIX_M * MATRIX_K * sizeof(half)));
	cudaErrCheck(hipMalloc((void**)&b_fp16, MATRIX_K * MATRIX_N * sizeof(half)));

	cudaErrCheck(hipMalloc((void**)&c, MATRIX_M * MATRIX_N * sizeof(float)));
	cudaErrCheck(hipMalloc((void**)&c_cublas, MATRIX_M * MATRIX_N * sizeof(float)));
	cudaErrCheck(hipMalloc((void**)&c_wmma, MATRIX_M * MATRIX_N * sizeof(float)));

	c_host_cublas = (float*)malloc(MATRIX_M * MATRIX_N * sizeof(float));
	c_host_wmma = (float*)malloc(MATRIX_M * MATRIX_N * sizeof(float));

	curandErrCheck(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
	curandErrCheck(hiprandSetPseudoRandomGeneratorSeed(gen, 1337ULL));

	curandErrCheck(hiprandGenerateUniform(gen, a_fp32, MATRIX_M * MATRIX_K));
	curandErrCheck(hiprandGenerateUniform(gen, b_fp32, MATRIX_K * MATRIX_N));

	// curand doesn't currently support fp16 so we generate in fp32 and convert to fp16.
	convertFp32ToFp16 <<< (MATRIX_M * MATRIX_K + 255) / 256, 256 >>> (a_fp16, a_fp32, MATRIX_M * MATRIX_K);
	convertFp32ToFp16 <<< (MATRIX_K * MATRIX_N + 255) / 256, 256 >>> (b_fp16, b_fp32, MATRIX_K * MATRIX_N);

	curandErrCheck(hiprandGenerateUniform(gen, c, MATRIX_M * MATRIX_N));

	curandErrCheck(hiprandDestroyGenerator(gen));

	cudaErrCheck(hipMemcpy(c_cublas, c, MATRIX_M * MATRIX_N * sizeof(float), hipMemcpyDeviceToDevice));
	cudaErrCheck(hipMemcpy(c_wmma, c, MATRIX_M * MATRIX_N * sizeof(float), hipMemcpyDeviceToDevice));

	float alpha = 1.0f;
	float beta = 0.0f;


	printf("\nM = %d, N = %d, K = %d. alpha = %f, beta = %f\n\n", MATRIX_M, MATRIX_N, MATRIX_K, alpha, beta);

	// First: using WMMA
	dim3 gridDim;
	dim3 blockDim;

	// blockDim.x must be a multple of warpSize
	// 128x4 means we have 16 warps and a block computes a 64x64 output tile
	blockDim.x = 16;
	blockDim.y = MATRIX_M;

	gridDim.x = (MATRIX_N + blockDim.x-1) / blockDim.x;
	gridDim.y = 1;

	printf("Running with wmma...\n");
	cudaErrCheck(hipEventRecord(startWMMA));
	wmma_example <<< gridDim, blockDim, 16*16*2*8 >>> (a_fp16, b_fp16, c_wmma, 16, 16, 16, alpha, beta);
	cudaErrCheck(hipEventRecord(stopWMMA));



	// Now using cuBLAS
	printf("Running with cuBLAS...\n");
	cudaErrCheck(hipEventRecord(startcublas));
	cublasErrCheck(hipblasGemmEx(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
				MATRIX_M, MATRIX_N, MATRIX_K, 
				&alpha,
				a_fp16, HIP_R_16F, MATRIX_M,
				b_fp16, HIP_R_16F, MATRIX_K,
				&beta, 
				c_cublas, HIP_R_32F, MATRIX_M,
				HIP_R_32F, CUBLAS_GEMM_DFALT_TENSOR_OP));
	cudaErrCheck(hipEventRecord(stopcublas));

	// Error checking
	printf("\nChecking results...\n");
	cudaErrCheck(hipMemcpy(c_host_wmma, c_wmma, MATRIX_M * MATRIX_N * sizeof(float), hipMemcpyDeviceToHost));
	cudaErrCheck(hipMemcpy(c_host_cublas, c_cublas, MATRIX_M * MATRIX_N * sizeof(float), hipMemcpyDeviceToHost));

	// 0.01% relative tolerance. 1e-5 absolute tolerance.
	int errors = 0;
	for (int i = 0; i < MATRIX_M * MATRIX_N; i++) {
		float v1 = c_host_wmma[i];
		float v2 = c_host_cublas[i];
		if (v1 / v2 > 1.01 || v2 / v1 > 1.01 || abs(v1 - v2) > 1e-2) {
			errors++;
			if (errors < 300) printf("%06d %f %f\n", i, v1, v2);
		}
	}

	if (errors > 0) {
		printf("WMMA does not agree with cuBLAS! %d errors!\n", errors);
	}
	else {
		printf("Results verified: cublas and WMMA agree.\n\n");
		float wmmaTime;
		float cublasTime;
		cudaErrCheck(hipEventSynchronize(stopWMMA));
		cudaErrCheck(hipEventSynchronize(stopcublas));
		cudaErrCheck(hipEventElapsedTime(&wmmaTime, startWMMA, stopWMMA));
		cudaErrCheck(hipEventElapsedTime(&cublasTime, startcublas, stopcublas));
		printf("wmma took %fms\n", wmmaTime);
		printf("cublas took %fms\n", cublasTime);

		printf("\nFor a faster code using wmma you should check out the cudaTensorCoreGemm sample in the CUDA Toolkit.\nThis code was written as a demo only!\n\n");
	}


	cudaErrCheck(hipEventDestroy(startWMMA));
	cudaErrCheck(hipEventDestroy(stopWMMA));

	cudaErrCheck(hipEventDestroy(startcublas));             
	cudaErrCheck(hipEventDestroy(stopcublas));

	cudaErrCheck(hipFree(a_fp32));
	cudaErrCheck(hipFree(b_fp32));
	cudaErrCheck(hipFree(a_fp16));
	cudaErrCheck(hipFree(b_fp16));

	cudaErrCheck(hipFree(c));
	cudaErrCheck(hipFree(c_cublas));
	cudaErrCheck(hipFree(c_wmma));

	free(c_host_cublas);
	free(c_host_wmma);

	cudaErrCheck(hipDeviceReset());
	return 0;
}


